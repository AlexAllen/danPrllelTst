#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include "randgen.cu"
using namespace std;

const unsigned long int goes = 10000;
const unsigned long int threads = 256;
const unsigned long int blocks = 1280;

__global__ void kernel(unsigned long *hits);

int main(void)
{
	unsigned long hits[blocks*threads], *dev_hits, totalHits=0;
	double pi;

	hipMalloc((void**) &dev_hits, threads*blocks*sizeof(unsigned long));

	kernel<<<blocks, threads>>>(dev_hits);

	hipMemcpy(&hits, dev_hits, threads*blocks*sizeof(unsigned long), hipMemcpyDeviceToHost);
	hipFree(dev_hits);

	for(int i=0; i<blocks*threads; i++)
	{
		totalHits += hits[i];
	}

	pi = 4.0 * (double) totalHits / (double) (goes *blocks *threads);

	cout.precision(8);
	cout << "\nCudaPi\n";
	cout << "Random points used:  " << goes*blocks*threads << endl;
	cout << "Pi was calculated as: " << pi << endl;
	cout << "Cmath reckons pi is:  " << M_PI << "\n\n";


	return 0;
}

__global__ void kernel(unsigned long *hits)
{
	float x, y;
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned long state[N];
	int mti = N+1;

	hits[id]=0;

	setSeed(0x8C*id-12, state, mti);

	for(int j=0; j < goes; j++)
	{
		x = rnd(state, mti);
		y = rnd(state, mti);

		if(x*x+y*y <= 1) hits[id]++;
	}
}

